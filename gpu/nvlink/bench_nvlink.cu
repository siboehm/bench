#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <sstream>
#include <string>
#include <thread>
#include <vector>

#define CHECK(call)                                                            \
  {                                                                            \
    const hipError_t error = call;                                            \
    if (error != hipSuccess) {                                                \
      std::cout << "Error: " << __FILE__ << ":" << __LINE__ << ", ";           \
      std::cout << "code: " << error                                           \
                << ", reason: " << hipGetErrorString(error);                  \
      std::cout << std::endl;                                                  \
      exit(1);                                                                 \
    }                                                                          \
  }

std::string bytes_to_human_readable(size_t size) {
  static const char *sizes[] = {"B", "KB", "MB", "GB", "TB"};
  int order = 0;
  double size_d = (double)size;
  while (size_d >= 1000 && order < 4) {
    size_d /= 1000;
    ++order;
  }

  std::stringstream ss;
  ss.precision(1);
  ss << std::fixed << size_d << sizes[order];

  return ss.str();
}

template <typename T> void printArray(T arr[], int size) {
  for (int i = 0; i < size; i++) {
    std::cout << arr[i] << " ";
  }
  std::cout << std::endl;
}

void storeTimingsToFile(const std::vector<std::chrono::nanoseconds> &timeData,
                        const std::vector<size_t> sizes,
                        const std::string filename) {
  std::ofstream file;
  file.open(filename);
  file << "Size,Time\n";
  for (int i = 0; i < timeData.size(); i++) {
    file << sizes[i] << "," << timeData[i].count() << "\n";
  }
  file.close();
}

const size_t WARMUP_STEPS = 2;
const size_t WARMUP_SIZE = 32 * 1024 * 1024; // 134 MB

const std::vector<size_t> SIZES = {
    8,        16,       32,       64,        128,      256,     512,
    1024,     2048,     4096,     8192,      16384,    32768,   65536,
    131072,   262144,   524288,   1048576,   2097152,  4194304, 8388608,
    16777216, 33554432, 67108864, 134217728, 268435456};

void checkTransfer(float *d0_data0, float *d0_data1, float *d1_data0,
                   float *d1_data1, hipStream_t d0_stream,
                   hipStream_t d1_stream) {

  // ensure pointers are on the correct device
  hipPointerAttribute_t d0_data0_attr;
  hipPointerAttribute_t d0_data1_attr;
  hipPointerAttribute_t d1_data0_attr;
  hipPointerAttribute_t d1_data1_attr;

  CHECK(hipPointerGetAttributes(&d0_data0_attr, d0_data0));
  CHECK(hipPointerGetAttributes(&d0_data1_attr, d0_data1));
  CHECK(hipPointerGetAttributes(&d1_data0_attr, d1_data0));
  CHECK(hipPointerGetAttributes(&d1_data1_attr, d1_data1));

  if (d0_data0_attr.device != 0 || d0_data1_attr.device != 0 ||
      d1_data0_attr.device != 1 || d1_data1_attr.device != 1) {
    throw std::runtime_error("Incorrect device for pointer");
  }

  std::vector<float> hsrc_d0(WARMUP_SIZE);
  std::vector<float> hsrc_d1(WARMUP_SIZE);
  std::vector<float> hdst_d0(WARMUP_SIZE);
  std::vector<float> hdst_d1(WARMUP_SIZE);

  for (int i = 0; i < hsrc_d0.size(); i++) {
    hsrc_d0[i] = (float)WARMUP_SIZE - i;
    hsrc_d1[i] = i + 1.0f;
  }

  CHECK(hipMemcpy(d0_data0, hsrc_d0.data(), WARMUP_SIZE * sizeof(float),
                   hipMemcpyDefault));
  CHECK(hipMemcpy(d1_data0, hsrc_d1.data(), WARMUP_SIZE * sizeof(float),
                   hipMemcpyDefault));

  CHECK(hipMemcpyPeerAsync(d1_data1, 1, d0_data0, 0,
                            WARMUP_SIZE * sizeof(float), d0_stream));
  CHECK(hipMemcpyPeerAsync(d0_data1, 0, d1_data0, 1,
                            WARMUP_SIZE * sizeof(float), d1_stream));
  CHECK(hipStreamSynchronize(d0_stream));
  CHECK(hipStreamSynchronize(d1_stream));

  CHECK(hipSetDevice(0));
  CHECK(hipMemcpy(hdst_d0.data(), d0_data1, WARMUP_SIZE * sizeof(float),
                   hipMemcpyDeviceToHost));

  CHECK(hipSetDevice(1));
  CHECK(hipMemcpy(hdst_d1.data(), d1_data1, WARMUP_SIZE * sizeof(float),
                   hipMemcpyDeviceToHost));

  for (int i = 0; i < WARMUP_SIZE; i++) {
    if (hdst_d1[i] != hsrc_d0[i]) {
      std::ostringstream ss;
      ss << "Divergence! hdst_d1[" << i << "]=" << hdst_d1[i] << " hsrc_d0["
         << i << "]=" << hsrc_d0[i] << std::endl;
      throw std::runtime_error(ss.str());
    }
    if (hdst_d0[i] != hsrc_d1[i]) {
      std::ostringstream ss;
      ss << "Divergence! hdst_d0[" << i << "]=" << hdst_d0[i] << " hsrc_d1["
         << i << "]=" << hsrc_d1[i] << std::endl;
      throw std::runtime_error(ss.str());
    }
  }
  std::cout << "Transfer check passed" << std::endl;
}

int main() {
  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  if (deviceCount < 2) {
    std::cout << "This benchmark requires at least two devices" << std::endl;
    return 1;
  }

  CHECK(hipSetDevice(0));
  hipStream_t d0_stream;
  hipStreamCreate(&d0_stream);
  float *d0_data0, *d0_data1;
  CHECK(hipMalloc(&d0_data0, SIZES.back() * sizeof(float)));
  CHECK(hipMalloc(&d0_data1, SIZES.back() * sizeof(float)));
  CHECK(hipDeviceEnablePeerAccess(1, 0)); // needs to be called on both devices

  CHECK(hipSetDevice(1));
  hipStream_t d1_stream;
  hipStreamCreate(&d1_stream);
  float *d1_data0, *d1_data1;
  CHECK(hipMalloc(&d1_data0, SIZES.back() * sizeof(float)));
  CHECK(hipMalloc(&d1_data1, SIZES.back() * sizeof(float)));
  CHECK(hipDeviceEnablePeerAccess(0, 0));

  checkTransfer(d0_data0, d0_data1, d1_data0, d1_data1, d0_stream, d1_stream);

  std::vector<std::chrono::nanoseconds> timeData;

  for (int i = 0; i < WARMUP_STEPS; i++) {
    CHECK(hipMemcpyPeerAsync(d1_data1, 1, d0_data0, 0,
                              WARMUP_SIZE * sizeof(float), d0_stream));
    CHECK(hipMemcpyPeerAsync(d0_data0, 0, d1_data1, 1,
                              WARMUP_SIZE * sizeof(float), d1_stream));
  }
  CHECK(hipStreamSynchronize(d0_stream));
  CHECK(hipStreamSynchronize(d1_stream));

  for (auto size : SIZES) {
    size_t sizeBytes = size * sizeof(float);
    auto ts = std::chrono::system_clock::now();
    CHECK(hipMemcpyPeerAsync(d1_data1, 1, d0_data0, 0, sizeBytes, d0_stream));
    CHECK(hipMemcpyPeerAsync(d0_data1, 0, d1_data0, 1, sizeBytes, d1_stream));
    CHECK(hipStreamSynchronize(d0_stream));
    CHECK(hipStreamSynchronize(d1_stream));
    timeData.push_back(std::chrono::system_clock::now() - ts);

    std::cout << "Size: " << bytes_to_human_readable(sizeBytes)
              << " Time: " << timeData.back().count() / 1000 << "μs"
              << " BW (duplex): "
              << ((double)sizeBytes * 1024) / timeData.back().count()
              << " MB/s\n";
  }

  storeTimingsToFile(timeData, SIZES, "build/cudaMemcpyPeerAsync.csv");

  CHECK(hipGetLastError());

  hipFree(d0_data0);
  hipFree(d0_data1);
  hipFree(d1_data0);
  hipFree(d1_data1);
  hipStreamDestroy(d0_stream);
  hipStreamDestroy(d1_stream);

  return 0;
}