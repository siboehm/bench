#include <chrono>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <sstream>
#include <string>
#include <thread>
#include <vector>

#define CUDA_CHECK(call)                                                       \
  {                                                                            \
    const hipError_t error = call;                                            \
    if (error != hipSuccess) {                                                \
      std::cout << "Error: " << __FILE__ << ":" << __LINE__ << ", ";           \
      std::cout << "code: " << error                                           \
                << ", reason: " << hipGetErrorString(error);                  \
      std::cout << std::endl;                                                  \
      exit(1);                                                                 \
    }                                                                          \
  }

std::string bytes_to_human_readable(size_t size) {
  static const char *sizes[] = {"B", "KB", "MB", "GB", "TB"};
  int order = 0;
  double size_d = (double)size;
  while (size_d >= 1000 && order < 4) {
    size_d /= 1000;
    ++order;
  }

  std::stringstream ss;
  ss.precision(1);
  ss << std::fixed << size_d << sizes[order];

  return ss.str();
}

void runPcieDefault(int size, std::vector<std::chrono::nanoseconds> &time_data,
                    hipStream_t stream_in, hipStream_t stream_out,
                    bool verbose = true) {
  int sizeBytes = size * sizeof(float);

  float *in_host, *out_host;
  in_host = (float *)malloc(sizeBytes);
  out_host = (float *)malloc(sizeBytes);
  float *in_dev, *out_dev;
  CUDA_CHECK(hipMalloc((void **)&in_dev, sizeBytes));
  CUDA_CHECK(hipMalloc((void **)&out_dev, sizeBytes));

  auto ts = std::chrono::system_clock::now();

  CUDA_CHECK(hipMemcpyAsync((void **)in_host, out_dev, sizeBytes,
                             hipMemcpyDeviceToHost, stream_in));
  CUDA_CHECK(hipMemcpyAsync((void **)in_dev, out_host, sizeBytes,
                             hipMemcpyHostToDevice, stream_out));
  CUDA_CHECK(hipStreamSynchronize(stream_in));
  CUDA_CHECK(hipStreamSynchronize(stream_out));

  time_data.push_back(std::chrono::system_clock::now() - ts);
  if (verbose) {
    std::cout << "Size: " << bytes_to_human_readable(sizeBytes)
              << " Time: " << time_data.back().count() / 1000 << "μs"
              << " BW (Dev to Host): "
              << ((double)sizeBytes * 1024) / time_data.back().count()
              << " MB/s" << std::endl;
  }

  free(out_host);
  hipFree(out_dev);
}

void runPciePinned(int size, std::vector<std::chrono::nanoseconds> &time_data,
                   hipStream_t stream_in, hipStream_t stream_out) {
  long sizeBytes = size * sizeof(float);

  float *in_host, *out_host;
  CUDA_CHECK(
      hipHostAlloc((void **)&out_host, sizeBytes, hipHostMallocDefault));
  CUDA_CHECK(hipHostAlloc((void **)&in_host, sizeBytes, hipHostMallocDefault));
  float *in_dev, *out_dev;
  CUDA_CHECK(hipMalloc((void **)&in_dev, sizeBytes));
  CUDA_CHECK(hipMalloc((void **)&out_dev, sizeBytes));

  auto ts = std::chrono::system_clock::now();

  CUDA_CHECK(hipMemcpyAsync((void **)in_host, out_dev, sizeBytes,
                             hipMemcpyDeviceToHost, stream_in));
  CUDA_CHECK(hipMemcpyAsync((void **)in_dev, out_host, sizeBytes,
                             hipMemcpyHostToDevice, stream_out));
  CUDA_CHECK(hipStreamSynchronize(stream_in));
  CUDA_CHECK(hipStreamSynchronize(stream_out));

  time_data.push_back(std::chrono::system_clock::now() - ts);
  std::cout << "Size: " << bytes_to_human_readable(sizeBytes)
            << " Time: " << time_data.back().count() / 1000 << "μs"
            << " BW (each direction): "
            << ((double)sizeBytes * 1000) / time_data.back().count() << " MB/s"
            << std::endl;

  hipHostFree(out_host);
  hipFree(out_dev);
}

void runPciePinnedWriteCombined(
    int size, std::vector<std::chrono::nanoseconds> &time_data,
    hipStream_t stream_in, hipStream_t stream_out) {
  long sizeBytes = size * sizeof(float);

  float *in_host, *out_host;
  CUDA_CHECK(
      hipHostAlloc((void **)&out_host, sizeBytes, hipHostMallocWriteCombined));
  CUDA_CHECK(
      hipHostAlloc((void **)&in_host, sizeBytes, hipHostMallocWriteCombined));
  float *in_dev, *out_dev;
  CUDA_CHECK(hipMalloc((void **)&in_dev, sizeBytes));
  CUDA_CHECK(hipMalloc((void **)&out_dev, sizeBytes));

  auto ts = std::chrono::system_clock::now();

  CUDA_CHECK(hipMemcpyAsync((void **)in_host, out_dev, sizeBytes,
                             hipMemcpyDeviceToHost, stream_in));
  CUDA_CHECK(hipMemcpyAsync((void **)in_dev, out_host, sizeBytes,
                             hipMemcpyHostToDevice, stream_out));
  CUDA_CHECK(hipStreamSynchronize(stream_in));
  CUDA_CHECK(hipStreamSynchronize(stream_out));

  time_data.push_back(std::chrono::system_clock::now() - ts);
  std::cout << "Size: " << bytes_to_human_readable(sizeBytes)
            << " Time: " << time_data.back().count() / 1000 << "μs"
            << " BW (Dev to Host): "
            << ((double)sizeBytes * 1024) / time_data.back().count() << " MB/s"
            << std::endl;

  hipHostFree(out_host);
  hipFree(out_dev);
}

void storeTimingsToFile(const std::vector<std::chrono::nanoseconds> &time_data,
                        const std::vector<size_t> sizes,
                        const std::string filename) {
  std::ofstream file;
  file.open(filename);
  file << "Size,Time\n";
  for (int i = 0; i < time_data.size(); i++) {
    file << sizes[i] << "," << time_data[i].count() << "\n";
  }
  file.close();
}

const size_t WARMUP = 2 << 14;

int main() {
  int deviceId;
  hipGetDevice(&deviceId);
  hipDeviceProp_t props{};
  hipGetDeviceProperties(&props, deviceId);
  std::cout << "Device: " << props.name << "\n";
  if (props.asyncEngineCount < 2) {
    std::cout << "No duplex Pcie" << std::endl;
    return 0;
  }

  std::vector<std::chrono::nanoseconds> warmup_data;
  std::vector<std::chrono::nanoseconds> time_data;

  std::vector<size_t> SIZES = {
      8,        16,       32,       64,        128,      256,     512,
      1024,     2048,     4096,     8192,      16384,    32768,   65536,
      131072,   262144,   524288,   1048576,   2097152,  4194304, 8388608,
      16777216, 33554432, 67108864, 134217728, 268435456};

  hipStream_t stream_in, stream_out;
  hipStreamCreate(&stream_in);
  hipStreamCreate(&stream_out);

  std::cout << "------- Running PCIe Normal ------"
            << "\n\n";
  // warmup
  runPcieDefault(WARMUP, warmup_data, stream_in, stream_out, false);
  for (auto size : SIZES) {
    runPcieDefault(size, time_data, stream_in, stream_out);
  }
  storeTimingsToFile(time_data, SIZES, "../pcie_normal.csv");
  time_data.clear();

  // sleep
  std::this_thread::sleep_for(std::chrono::seconds(5));

  std::cout << "\n----- Running PCIe Pinned ------"
            << "\n\n";
  // warmup
  runPcieDefault(WARMUP, warmup_data, stream_in, stream_out, false);
  for (auto size : SIZES) {
    runPciePinned(size, time_data, stream_in, stream_out);
  }
  storeTimingsToFile(time_data, SIZES, "../pcie_pinned.csv");
  time_data.clear();

  std::this_thread::sleep_for(std::chrono::seconds(5));

  std::cout << "\n----- Running PCIe Pinned Write Combined ------"
            << "\n\n";
  // warmup
  runPcieDefault(WARMUP, warmup_data, stream_in, stream_out, false);
  for (auto size : SIZES) {
    runPciePinnedWriteCombined(size, time_data, stream_in, stream_out);
  }
  storeTimingsToFile(time_data, SIZES, "../pcie_pinned_write_combined.csv");
  time_data.clear();

  hipStreamDestroy(stream_in);
  hipStreamDestroy(stream_out);
}