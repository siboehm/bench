#include <chrono>
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

int main() {

  std::vector<std::chrono::duration<double>> time_data;

  auto ts = std::chrono::system_clock::now();

  hipStream_t stream_in, stream_out;
  hipStreamCreate(&stream_in);
  hipStreamCreate(&stream_out);

  int size = 1000 * 1000 * 1000;
  int *in_host = (int *)malloc(1000 * size);
  int *out_host = (int *)malloc(1000 * size);
  int *in_dev, *out_dev;
  hipMalloc((void **)&in_dev, 1000 * size);
  hipMalloc((void **)&out_dev, 1000 * size);

  hipMemcpyAsync((void **)in_dev, in_host, 1000 * size, hipMemcpyHostToDevice,
                  stream_in);
  hipMemcpyAsync((void **)out_host, out_dev, 1000 * size,
                  hipMemcpyDeviceToHost, stream_out);

  hipStreamSynchronize(stream_in);
  hipStreamSynchronize(stream_out);

  time_data.push_back(std::chrono::system_clock::now() - ts);
  std::cout << "Time taken: " << time_data.back().count() << "s " << std::endl;
}