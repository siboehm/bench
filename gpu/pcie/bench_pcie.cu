#include <chrono>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <sstream>
#include <string>
#include <thread>
#include <vector>

#define CUDA_CHECK(call)                                                       \
  {                                                                            \
    const hipError_t error = call;                                            \
    if (error != hipSuccess) {                                                \
      std::cout << "Error: " << __FILE__ << ":" << __LINE__ << ", ";           \
      std::cout << "code: " << error                                           \
                << ", reason: " << hipGetErrorString(error);                  \
      std::cout << std::endl;                                                  \
      exit(1);                                                                 \
    }                                                                          \
  }

std::string bytes_to_human_readable(size_t size) {
  static const char *sizes[] = {"B", "KB", "MB", "GB", "TB"};
  int order = 0;
  double size_d = (double)size;
  while (size_d >= 1024 && order < 4) {
    size_d /= 1024;
    ++order;
  }

  std::stringstream ss;
  ss.precision(2);
  ss << std::fixed << size_d << sizes[order];

  return ss.str();
}

void runPcieNormal(int size, std::vector<std::chrono::nanoseconds> &time_data,
                   hipStream_t stream_in, hipStream_t stream_out) {
  int sizeBytes = size * sizeof(float);

  float *in_host = (float *)malloc(sizeBytes);
  float *out_host = (float *)malloc(sizeBytes);
  float *in_dev, *out_dev;
  CUDA_CHECK(hipMalloc((void **)&in_dev, sizeBytes));
  CUDA_CHECK(hipMalloc((void **)&out_dev, sizeBytes));

  auto ts = std::chrono::system_clock::now();

  CUDA_CHECK(hipMemcpyAsync((void **)in_dev, in_host, sizeBytes,
                             hipMemcpyHostToDevice, stream_in));
  CUDA_CHECK(hipMemcpyAsync((void **)out_host, out_dev, sizeBytes,
                             hipMemcpyDeviceToHost, stream_out));
  CUDA_CHECK(hipStreamSynchronize(stream_in));
  CUDA_CHECK(hipStreamSynchronize(stream_out));

  time_data.push_back(std::chrono::system_clock::now() - ts);
  std::cout << "Size: " << bytes_to_human_readable(sizeBytes)
            << " Time: " << time_data.back().count() / 1000 << "μs"
            << " Effective BW: "
            << ((double)sizeBytes * 1024) / time_data.back().count() << " MB/s"
            << std::endl;

  free(in_host);
  free(out_host);
  hipFree(in_dev);
  hipFree(out_dev);
}

void runPciePinned(int size, std::vector<std::chrono::nanoseconds> &time_data,
                   hipStream_t stream_in, hipStream_t stream_out) {
  long sizeBytes = size * sizeof(float);

  float *in_host, *out_host;
  CUDA_CHECK(hipHostMalloc((void **)&in_host, sizeBytes, hipHostMallocDefault));
  CUDA_CHECK(hipHostMalloc((void **)&out_host, sizeBytes, hipHostMallocDefault));
  float *in_dev, *out_dev;
  CUDA_CHECK(hipMalloc((void **)&in_dev, sizeBytes));
  CUDA_CHECK(hipMalloc((void **)&out_dev, sizeBytes));

  auto ts = std::chrono::system_clock::now();

  CUDA_CHECK(hipMemcpyAsync((void **)in_dev, in_host, sizeBytes,
                             hipMemcpyHostToDevice, stream_in));
  CUDA_CHECK(hipMemcpyAsync((void **)out_host, out_dev, sizeBytes,
                             hipMemcpyDeviceToHost, stream_out));
  CUDA_CHECK(hipStreamSynchronize(stream_in));
  CUDA_CHECK(hipStreamSynchronize(stream_out));

  time_data.push_back(std::chrono::system_clock::now() - ts);
  std::cout << "Size: " << bytes_to_human_readable(sizeBytes)
            << " Time: " << time_data.back().count() / 1000 << "μs"
            << " Effective BW: "
            << ((double)sizeBytes * 1024) / time_data.back().count() << " MB/s"
            << std::endl;

  hipHostFree(in_host);
  hipHostFree(out_host);
  hipFree(in_dev);
  hipFree(out_dev);
}

void runPciePinnedWriteCombined(
    int size, std::vector<std::chrono::nanoseconds> &time_data,
    hipStream_t stream_in, hipStream_t stream_out) {
  long sizeBytes = size * sizeof(float);

  float *in_host, *out_host;
  CUDA_CHECK(
      hipHostMalloc((void **)&in_host, sizeBytes, hipHostMallocWriteCombined, hipHostMallocDefault));
  CUDA_CHECK(hipHostMalloc((void **)&out_host, sizeBytes,
                            hipHostMallocWriteCombined, hipHostMallocDefault));
  float *in_dev, *out_dev;
  CUDA_CHECK(hipMalloc((void **)&in_dev, sizeBytes));
  CUDA_CHECK(hipMalloc((void **)&out_dev, sizeBytes));

  auto ts = std::chrono::system_clock::now();

  CUDA_CHECK(hipMemcpyAsync((void **)in_dev, in_host, sizeBytes,
                             hipMemcpyHostToDevice, stream_in));
  CUDA_CHECK(hipMemcpyAsync((void **)out_host, out_dev, sizeBytes,
                             hipMemcpyDeviceToHost, stream_out));
  CUDA_CHECK(hipStreamSynchronize(stream_in));
  CUDA_CHECK(hipStreamSynchronize(stream_out));

  time_data.push_back(std::chrono::system_clock::now() - ts);
  std::cout << "Size: " << bytes_to_human_readable(sizeBytes)
            << " Time: " << time_data.back().count() / 1000 << "μs"
            << " Effective BW: "
            << ((double)sizeBytes * 1024) / time_data.back().count() << " MB/s"
            << std::endl;

  hipHostFree(in_host);
  hipHostFree(out_host);
  hipFree(in_dev);
  hipFree(out_dev);
}

void storeTimingsToFile(const std::vector<std::chrono::nanoseconds> &time_data,
                        const std::vector<int> sizes,
                        const std::string filename) {
  std::ofstream file;
  file.open(filename);
  file << "Size,Time\n";
  for (int i = 0; i < time_data.size(); i++) {
    file << sizes[i] << "," << time_data[i].count() << "\n";
  }
  file.close();
}

int main() {

  std::vector<std::chrono::nanoseconds> time_data;

  std::vector<int> SIZES = {1,        2,        4,         8,        16,
                            32,       64,       128,       256,      512,
                            1024,     2048,     4096,      8192,     16384,
                            32768,    65536,    131072,    262144,   524288,
                            1048576,  2097152,  4194304,   8388608,  16777216,
                            33554432, 67108864, 134217728, 268435456};

  hipStream_t stream_in, stream_out;
  hipStreamCreate(&stream_in);
  hipStreamCreate(&stream_out);

  std::cout << "------- Running PCIe Normal ------"
            << "\n\n";
  for (auto size : SIZES) {
    runPcieNormal(size, time_data, stream_in, stream_out);
  }
  storeTimingsToFile(time_data, SIZES, "pcie_normal.csv");
  time_data.clear();

  std::cout << "\n----- Running PCIe Pinned ------"
            << "\n\n";
  for (auto size : SIZES) {
    runPciePinned(size, time_data, stream_in, stream_out);
  }
  storeTimingsToFile(time_data, SIZES, "pcie_pinned.csv");
  time_data.clear();

  std::cout << "\n----- Running PCIe Pinned Write Combined ------"
            << "\n\n";
  for (auto size : SIZES) {
    runPciePinnedWriteCombined(size, time_data, stream_in, stream_out);
  }
  storeTimingsToFile(time_data, SIZES, "pcie_pinned_write_combined.csv");
  time_data.clear();

  hipStreamDestroy(stream_in);
  hipStreamDestroy(stream_out);
}